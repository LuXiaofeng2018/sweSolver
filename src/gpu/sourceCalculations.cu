
#include <hip/hip_runtime.h>
__global__ void bedSlopeSourceSolver(float *BedSlopeSource, float *U, float *BottomIntPts, int m, int n, float dx, float dy)
{
	// Calculate the row and column of the thread within the thread block
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	// First check if the thread is operating on a cell inside of the block's one cell deep ghost cells
	if (col > 0 && row > 0 && col < n-1 && row < m-1)
	{
		// Calculate index of this cell in the BedSlopeSource matrix
		int slopeIndex = row*n*2 + col*2;
		
		// Calculate index of this cell in the U matrix
		int uIndex = row*n*3 + col*3;
		
		// Calculate index of this cell in the BottomIntPts matrix
		int bottomIndex = row*(n+1)*2 + col*2;
		
		// Calculate the water depth at the center of the cell
		// Note: h = w - B;
		//	 w is the water column height at the cell center measured from z = 0;
		//	 B is the terrain elevation measured from z = 0, and is calculated as the average
		//	   elevation of the center of two opposing edges (in this case, left and right)
		float hCenter =  U[uIndex] - (BottomIntPts[bottomIndex+1] + BottomIntPts[bottomIndex + 1*2 + 1])/2.0f;
		
		// Calculate the slope of the terrain in both x- and y-directions
		float slopeX = (BottomIntPts[bottomIndex + 3] - BottomIntPts[bottomIndex + 1]) / dx;
		float slopeY = (BottomIntPts[bottomIndex + (n+1)*2] - BottomIntPts[bottomIndex]) / dy;
		
		// Calculate the bed slope source terms
		BedSlopeSource[slopeIndex] = -9.81f * slopeX * hCenter;
		BedSlopeSource[slopeIndex+1] = -9.81f * slopeY * hCenter;		 
	}
}
