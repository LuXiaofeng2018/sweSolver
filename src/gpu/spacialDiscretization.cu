
#include <hip/hip_runtime.h>
__device__ float minmod(float a, float b, float c)
{
	float ab = fminf(fabsf(a), fabs(b)) * (copysignf(1.0f, a) + copysignf(1.0f, b)) * 0.5f;
	return fminf(fabsf(ab), fabsf(c)) * (copysignf(1.0f, ab) + copysignf(1.0f, c)) * 0.5f;
}
    
    
__global__ void ReconstructFreeSurface(float *U, float *BottomIntPts, float *UIntPts, float *huvIntPts, int m, int n, float dx, float dy)
{
	// Calculate the row and column of the thread within the thread block
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Calculate the index of the up, down, left, right terrain elevation values in the BottomIntPts vector
	// 	Note: We only need elevation values from two point on opposite sides of the cell
	//	      because the bottom is piecewise-bilinear, which guarantees that the average
	// 	      of both sets of points on opposing sides of the cell is equal to the value
	//	      at the center of the cell.
	int upTerrainIndex = (row+1)*(n+1)*2 + col*2;
	int downTerrainIndex = row*(n+1)*2 + col*2;
	int leftTerrainIndex = row*(n+1)*2 + col*2 + 1;
	int rightTerrainIndex = row*(n+1)*2 + (col+1)*2 + 1;
	
	// Before performing calculations for a timestep, we need to reset any residual values left over
	// from previous calculations in case a cell goes from wet to dry. In the case of a cell going
	// from wet to dry, the values of huvIntPts and UIntPts for that cell would otherwise remain at the 
	// values from the previous timestep when the cell was wet, causing errors.
	// 	huvIntPts: set h = u = v = 0
	//	UIntPts: set w = ground elevation, u = v = 0
	int cellIntPtIndex = row*n*4*3 + col*4*3;
	
	if (row < m && col < n)
	{
		for (int i=0; i<4; i++)
		{
			for (int j=0; j<3; j++)
			{
				huvIntPts[cellIntPtIndex + i*3 + j] = 0.0f;
				if (j > 0)
				{
					UIntPts[cellIntPtIndex + i*3 + j] = 0.0f;
				}
			}
		}
	
		UIntPts[cellIntPtIndex + 0*3] = BottomIntPts[upTerrainIndex];
		UIntPts[cellIntPtIndex + 1*3] = BottomIntPts[downTerrainIndex];
		UIntPts[cellIntPtIndex + 2*3] = BottomIntPts[rightTerrainIndex];
		UIntPts[cellIntPtIndex + 3*3] = BottomIntPts[leftTerrainIndex];
	}

	// First check if the thread is operating on a cell inside of the block's one cell deep ghost cells
	if (col > 0 && row > 0 && col < n-1 && row < m-1)
	{
	
		// Calculate the index of the cell in the U vector
		int wIndex = row*n*3 + col*3 + 0;
	
		// Now check if the cell has water (evaluates to true if it does)
		// TODO: It may be necessary to have an else statement as well. If a cell goes
		//	 from wet to dry, the value of w may drop below the ground level, so
		//	 if the cell needs to get wet again, the water level will have to increase
		//	 from that value, not from ground level
		if (U[wIndex] > (BottomIntPts[leftTerrainIndex] + BottomIntPts[rightTerrainIndex])/2.0f) 
		{
			float N[3], S[3], E[3], W[3]; // These are the [w, hu, hv] vectors
			float north[3], south[3], east[3], west[3]; // These are the [h, u, v] vectors
			
			float forward, central, backward, slope;
			float Kappa = 0.01f * fmaxf(1.0f, fminf(dx, dy));
			float sqrt2 = sqrtf(2.0f);
			
			// Calculate indices of cells surrounding the current cell in the U vector
			int upIndex = (row+1)*n*3 + col*3;
			int downIndex = (row-1)*n*3 + col*3;
			int leftIndex = row*n*3 + (col-1)*3;
			int rightIndex = row*n*3 + (col+1)*3;
			
			// Reconstruct the free surface so that it is sloped based on the free surface height of adjacent cells
			for (int i=0; i<3; i++)
			{
				// North and South
				forward = (U[upIndex+i] - U[wIndex+i])/dy;
				central = (U[upIndex+i] - U[downIndex+i])/(2.0f*dy);
				backward = (U[wIndex+i] - U[downIndex+i])/dy;
				slope = minmod(1.3f*forward, central, 1.3f*backward);
				
				N[i] = U[wIndex+i] + (dy/2.0f)*slope;
				S[i] = U[wIndex+i] - (dy/2.0f)*slope;
				
				// East and West
				forward = (U[rightIndex+i] - U[wIndex+i])/dx;
				central = (U[rightIndex+i] - U[leftIndex+i])/(2.0f*dx);
				backward = (U[wIndex+i] - U[leftIndex+i])/dx;
				slope = minmod(1.3f*forward, central, 1.3f*backward);
				
				E[i] = U[wIndex+i] + (dx/2.0f)*slope;
				W[i] = U[wIndex+i] - (dx/2.0f)*slope;
			}
			
			// Check the water free surface at the cell interfaces for positivity and correct if necessary
			if (N[0] < BottomIntPts[upTerrainIndex])
			{
				N[0] = BottomIntPts[upTerrainIndex];
				S[0] = 2*U[wIndex] - BottomIntPts[upTerrainIndex];
			}
			else if (S[0] < BottomIntPts[downTerrainIndex])
			{
				S[0] = BottomIntPts[downTerrainIndex];
				N[0] = 2*U[wIndex] - BottomIntPts[downTerrainIndex];
			}
			if (E[0] < BottomIntPts[rightTerrainIndex])
			{
				E[0] = BottomIntPts[rightTerrainIndex];
				W[0] = 2*U[wIndex] - BottomIntPts[rightTerrainIndex];
			}
			else if (W[0] < BottomIntPts[leftTerrainIndex])
			{
				W[0] = BottomIntPts[leftTerrainIndex];
				E[0] = 2*U[wIndex] - BottomIntPts[leftTerrainIndex];
			}
			
			// Calculate the values of h, u, and v using the damping calculations for u and v
			north[0] = N[0] - BottomIntPts[upTerrainIndex];
			south[0] = S[0] - BottomIntPts[downTerrainIndex];
			east[0] = E[0] - BottomIntPts[rightTerrainIndex];
			west[0] = W[0] - BottomIntPts[leftTerrainIndex];
			
			for (int i=1; i<3; i++)
			{
				north[i] = (sqrt2 * north[0] * N[i]) / sqrtf(powf(north[0], 4.0f) + fmaxf(powf(north[0], 4.0f), Kappa));
				south[i] = (sqrt2 * south[0] * S[i]) / sqrtf(powf(south[0], 4.0f) + fmaxf(powf(south[0], 4.0f), Kappa));
				east[i] = (sqrt2 * east[0] * E[i]) / sqrtf(powf(east[0], 4.0f) + fmaxf(powf(east[0], 4.0f), Kappa));
				west[i] = (sqrt2 * west[0] * W[i]) / sqrtf(powf(west[0], 4.0f) + fmaxf(powf(west[0], 4.0f), Kappa));
			}
			
			// Update the values of hu and hv based on new values of u and v
			for (int i=1; i<3; i++)
			{
				N[i] = north[0] * north[i];
				S[i] = south[0] * south[i];
				E[i] = east[0] * east[i];
				W[i] = west[0] * west[i];
			}
			
			// Put the calculated interface values into global memory
			for (int i=0; i<3; i++)
			{
				UIntPts[cellIntPtIndex + 0*3 + i] = N[i];
				UIntPts[cellIntPtIndex + 1*3 + i] = S[i];
				UIntPts[cellIntPtIndex + 2*3 + i] = E[i];
				UIntPts[cellIntPtIndex + 3*3 + i] = W[i];
				
				huvIntPts[cellIntPtIndex + 0*3 + i] = north[i];
				huvIntPts[cellIntPtIndex + 1*3 + i] = south[i];
				huvIntPts[cellIntPtIndex + 2*3 + i] = east[i];
				huvIntPts[cellIntPtIndex + 3*3 + i] = west[i];
			}
			
			// End the kernel here. The values of the conserved variable [h, u, v] need to be stored
			// in global memory for the entire domain before we can begin calculating propagation speeds.
		}
	}
}


__global__ void CalculatePropSpeeds(float *UIntPts, float *huvIntPts, float *propSpeeds, int m, int n)
{
	// Calculate the row and column of the thread within the thread block
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Constants
	float g = 9.81f;
	
	// Get rid of any residual values in the propSpeeds matrix in case
	// a cell has gone from wet to dry in the previous timestep
	for (int i=0; i<4; i++)
	{
		propSpeeds[row*n*4 + col*4 + i] = 0.0f;
	}
	
	// First check if the thread is operating on a cell inside of the block's one cell deep ghost cells
	if (col > 0 && row > 0 && col < n-1 && row < m-1)
	{
		
		// Make sure cell is wet by making sure at least one value of h is greater than 0
		float hNorth = huvIntPts[row*n*4*3 + col*4*3 + 0*3];		// value of h at the north interface of the current cell
		float hSouth = huvIntPts[(row+1)*n*4*3 + col*4*3 + 1*3];	// value of h at the south interface of the cell above
		float hEast = huvIntPts[row*n*4*3 + col*4*3 + 2*3];		// value of h at the east interface of the current cell
		float hWest = huvIntPts[row*n*4*3 + (col+1)*4*3 + 3*3];		// value of h at the west interface of the cell to the right
		if (hNorth > 0.0f || hSouth > 0.0f || hEast > 0.0f || hWest > 0.0f)
		{
			// Get the rest of the values needed from huvIntPts
			float vNorth = huvIntPts[row*n*4*3 + col*4*3 + 0*3 + 1];	// value of v at the north interface of the current cell
			float vSouth = huvIntPts[(row+1)*n*4*3 + col*4*3 + 1*3 + 1];	// value of v at the south interface of the cell above
			float uEast = huvIntPts[row*n*4*3 + col*4*3 + 2*3 + 1];		// value of u at the east interface of the current cell
			float uWest = huvIntPts[row*n*4*3 + (col+1)*4*3 + 3*3 + 1];	// value of u at the west interface of the cell to the right
		
			// Each cell in propSpeeds contains four values [N, S, E, W]
			// Each thread will calculate the N, E values of it's own cell, the S of the cell above, and the W of
			// the cell to the right.
			int N = row*n*4 + col*4 + 0;	// North value of this cell
			int S = (row+1)*n*4 + col*4 + 1;// South value of cell above
			int E = row*n*4 + col*4 + 2;	// East value of this cell
			int W = row*n*4 + (col+1)*4 + 3;// West value of cell to the right
		
			// Calculate north propagation speed of the current cell
			propSpeeds[N] = fminf(fminf(vNorth - sqrtf(g*hNorth), vSouth - sqrtf(g*hSouth)), 0.0f);
		
			// Calculate south propagation speed of the cell above
			propSpeeds[S] = fmaxf(fmaxf(vNorth + sqrtf(g*hNorth), vSouth + sqrtf(g*hSouth)), 0.0f);
		
			// Calculate east propagation speed of the current cell
			propSpeeds[E] = fminf(fminf(uEast - sqrtf(g*hEast), uWest - sqrtf(g*hWest)), 0.0f);
		
			// Calculate west propagation speed of the cell to the right
			propSpeeds[W] = fmaxf(fmaxf(uEast + sqrtf(g*hEast), uWest + sqrtf(g*hWest)), 0.0f);
		}
	}
}
